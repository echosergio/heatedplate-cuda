
#include <hip/hip_runtime.h>
#define M 500
#define N 500
#define NUM_ELEMENTS M *N
#define SHARED_MEMORY_ARRAY_SIZE 1024

#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

static void HandleError(hipError_t err, const char *file, int line)
{
    if (err != hipSuccess)
    {
        fprintf(stderr, "%s in %s at line %d\n", hipGetErrorString(err), file, line);
        exit(EXIT_FAILURE);
    }
}

#define CHECK_CUDA_ERROR(msg) (checkCUDAError(msg, __FILE__, __LINE__))

static void checkCUDAError(const char *msg, const char *file, int line)
{
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Cuda error: %s: %s. In %s at line %d\n", msg, hipGetErrorString(err), file, line);
        exit(EXIT_FAILURE);
    }
}

__global__ void copy_grid(double *d_w, double *d_u)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;

    if (x >= 0 && y >= 0 && x < M && y < N)
    {
        int index = x + y * N;

        d_u[index] = d_w[index];
        __syncthreads();
    }

    return;
}

__device__ double d_epsilon;

__device__ double d_epsilon_reduction_max[NUM_ELEMENTS];

__device__ int d_stride_shared_counter;

__global__ void epsilon_reduction(double *d_w, double *d_u)
{
    __shared__ double partial_epsilon_reduction_max[SHARED_MEMORY_ARRAY_SIZE];

    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;

    if (x >= 0 && y >= 0 && x < M && y < N)
    {
        int index = x + y * N;

        if (index == 0)
        {
            d_stride_shared_counter = NUM_ELEMENTS;
        }
        __syncthreads();

        d_epsilon_reduction_max[index] = fabs(d_w[index] - d_u[index]);
        __syncthreads();

        while (d_stride_shared_counter > SHARED_MEMORY_ARRAY_SIZE)
        {
            int local_index = index % SHARED_MEMORY_ARRAY_SIZE;
            partial_epsilon_reduction_max[local_index] = d_epsilon_reduction_max[index];
            __syncthreads();

            for (unsigned int stride = SHARED_MEMORY_ARRAY_SIZE / 2; stride > 0; stride >>= 1)
            {
                if (local_index < stride)
                    partial_epsilon_reduction_max[local_index] = max(partial_epsilon_reduction_max[local_index], partial_epsilon_reduction_max[local_index + stride]);
                __syncthreads();
            }

            if (local_index == 0)
            {
                int d_epsilon_reduction_max_index = index / SHARED_MEMORY_ARRAY_SIZE;
                d_epsilon_reduction_max[d_epsilon_reduction_max_index] = partial_epsilon_reduction_max[local_index];
            }

            if (index == 0)
            {
                d_stride_shared_counter = NUM_ELEMENTS / SHARED_MEMORY_ARRAY_SIZE;
            }
            __syncthreads();
        }

        int local_index = index % SHARED_MEMORY_ARRAY_SIZE;
            partial_epsilon_reduction_max[local_index] = d_epsilon_reduction_max[index];
        __syncthreads();

        for (unsigned int stride = d_stride_shared_counter / 2; stride > 0; stride >>= 1)
        {
            if (local_index < stride)
                partial_epsilon_reduction_max[local_index] = max(partial_epsilon_reduction_max[local_index], partial_epsilon_reduction_max[local_index + stride]);
            __syncthreads();
        }

        if (local_index == 0)
        {
            int d_epsilon_reduction_max_index = index / SHARED_MEMORY_ARRAY_SIZE;
            d_epsilon_reduction_max[d_epsilon_reduction_max_index] = partial_epsilon_reduction_max[local_index];
        }

        if (index == 0)
        {
            d_epsilon = d_epsilon_reduction_max[index];
        }
        __syncthreads();
    }

    return;
}

// CUDA kernel to perform the reduction in parallel on the GPU
//! @param g_idata  input data in global memory
//                  result is expected in index 0 of g_idata
//! @param n        input number of elements to scan from input data
__global__ void calculate_solution(double *d_w, double *d_u)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;

    if (x > 0 && y > 0 && x < M - 1 && y < N - 1)
    {
        int index = x + y * N;

        int left = (x - 1) + y * N;
        int right = (x + 1) + y * N;
        int top = x + (y - 1) * N;
        int bottom = x + (y + 1) * N;

        d_w[index] = (d_u[left] + d_u[right] + d_u[top] + d_u[bottom]) / 4.0;
        __syncthreads();
    }

    return;
}

void calculate_solution_kernel(double w[M][N], double epsilon)
{
    double diff;
    int iterations;
    int iterations_print;

    const unsigned int matrix_mem_size = sizeof(double) * M * N;

    double *d_w = (double *)malloc(matrix_mem_size);
    double *d_u = (double *)malloc(matrix_mem_size);

    // Memory allocation on device side
    HANDLE_ERROR(hipMalloc((void **)&d_w, matrix_mem_size));
    HANDLE_ERROR(hipMalloc((void **)&d_u, matrix_mem_size));

    // Copy from host memory to device memory
    HANDLE_ERROR(hipMemcpy(d_w, w, matrix_mem_size, hipMemcpyHostToDevice));

    // Dimensions for a 2D matrix with max size 512
    dim3 dimGrid(16, 16);  // 256 blocks
    dim3 dimBlock(32, 32); // 1024 threads

    diff = epsilon;

    iterations = 0;
    iterations_print = 1;
    printf("\n");
    printf(" Iteration  Change\n");
    printf("\n");

    while (epsilon <= diff)
    {
        copy_grid<<<dimGrid, dimBlock>>>(d_w, d_u);
        calculate_solution<<<dimGrid, dimBlock>>>(d_w, d_u);
        epsilon_reduction<<<dimGrid, dimBlock>>>(d_w, d_u);

        hipDeviceSynchronize();

        HANDLE_ERROR(hipMemcpyFromSymbol(&diff, HIP_SYMBOL(d_epsilon), sizeof(double), 0, hipMemcpyDeviceToHost));

        iterations++;
        if (iterations == iterations_print)
        {
            printf("  %8d  %lg\n", iterations, diff);
            iterations_print = 2 * iterations_print;
        }
    }

    CHECK_CUDA_ERROR("Kernel invocation");

    printf("\n");
    printf("  %8d  %lg\n", iterations, diff);
    printf("\n");
    printf("  Error tolerance achieved.\n");
    //printf("  CPU time = %f\n", ctime);

    // Copy from device memory back to host memory
    HANDLE_ERROR(hipMemcpy(w, d_w, matrix_mem_size, hipMemcpyDeviceToHost));

    hipFree(d_w);
    hipFree(d_u);
}

#undef M
#undef N
#undef NUM_ELEMENTS