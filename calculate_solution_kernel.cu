
#include <hip/hip_runtime.h>
#define M 64
#define N 64
#define NUM_ELEMENTS 4096

#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

static void HandleError(hipError_t err, const char *file, int line)
{
    if (err != hipSuccess)
    {
        fprintf(stderr, "%s in %s at line %d\n", hipGetErrorString(err), file, line);
        exit(EXIT_FAILURE);
    }
}

#define CHECK_CUDA_ERROR(msg) (checkCUDAError(msg, __FILE__, __LINE__))

static void checkCUDAError(const char *msg, const char *file, int line)
{
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Cuda error: %s: %s. In %s at line %d\n", msg, hipGetErrorString(err), file, line);
        exit(EXIT_FAILURE);
    }
}

__global__ void copy_grid(double *d_w, double *d_u)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;

    if (x >= 0 && y >= 0 && x < M && y < N)
    {
        int index = x + y * N;

        d_u[index] = d_w[index];
        __syncthreads();
    }

    return;
}

__device__ double d_epsilon;

__device__ double epsilon_reduction_max[NUM_ELEMENTS];

__global__ void epsilon_reduction(double *d_w, double *d_u)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;

    if (x >= 0 && y >= 0 && x < M && y < N)
    {
        int index = x + y * N;

        epsilon_reduction_max[index] = fabs(d_w[index] - d_u[index]);
        __syncthreads();

        for (unsigned int stride = NUM_ELEMENTS / 2; stride > 0; stride >>= 1)
        {
            if (index < stride)
                epsilon_reduction_max[index] = max(epsilon_reduction_max[index], epsilon_reduction_max[index + stride]); 
            __syncthreads();
        }
    }

    if (x == 0 && y == 0)
    {
        d_epsilon = epsilon_reduction_max[x + y * N];
    }

    return;
}

// CUDA kernel to perform the reduction in parallel on the GPU
//! @param g_idata  input data in global memory
//                  result is expected in index 0 of g_idata
//! @param n        input number of elements to scan from input data
__global__ void calculate_solution(double *d_w, double *d_u)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;

    if (x > 0 && y > 0 && x < M - 1 && y < N - 1)
    {
        int index = x + y * N;

        int left = (x - 1) + y * N;
        int right = (x + 1) + y * N;
        int top = x + (y - 1) * N;
        int bottom = x + (y + 1) * N;

        d_w[index] = (d_u[left] + d_u[right] + d_u[top] + d_u[bottom]) / 4.0;
        __syncthreads();
    }

    return;
}

void calculate_solution_kernel(double w[M][N], double epsilon)
{
    double diff;
    int iterations;
    int iterations_print;

    const unsigned int matrix_mem_size = sizeof(double) * M * N;

    double *d_w = (double *)malloc(matrix_mem_size);
    double *d_u = (double *)malloc(matrix_mem_size);

    // Memory allocation on device side
    HANDLE_ERROR(hipMalloc((void **)&d_w, matrix_mem_size));
    HANDLE_ERROR(hipMalloc((void **)&d_u, matrix_mem_size));

    // Copy from host memory to device memory
    HANDLE_ERROR(hipMemcpy(d_w, w, matrix_mem_size, hipMemcpyHostToDevice));

    // Dimensions for a 2D matrix with max size 512
    dim3 dimGrid(16, 16);  // 256 blocks
    dim3 dimBlock(32, 32); // 1024 threads

    diff = epsilon;

    iterations = 0;
    iterations_print = 1;
    printf("\n");
    printf(" Iteration  Change\n");
    printf("\n");

    while (epsilon <= diff)
    {
        copy_grid<<<dimGrid, dimBlock>>>(d_w, d_u);
        calculate_solution<<<dimGrid, dimBlock>>>(d_w, d_u);
        epsilon_reduction<<<dimGrid, dimBlock>>>(d_w, d_u);

        hipDeviceSynchronize();

        HANDLE_ERROR(hipMemcpyFromSymbol(&diff, HIP_SYMBOL(d_epsilon), sizeof(double), 0, hipMemcpyDeviceToHost));

        iterations++;
        if (iterations == iterations_print)
        {
            printf("  %8d  %lg\n", iterations, diff);
            iterations_print = 2 * iterations_print;
        }
    }

    CHECK_CUDA_ERROR("Kernel invocation");

    printf("\n");
    printf("  %8d  %lg\n", iterations, diff);
    printf("\n");
    printf("  Error tolerance achieved.\n");
    //printf("  CPU time = %f\n", ctime);

    // Copy from device memory back to host memory
    HANDLE_ERROR(hipMemcpy(w, d_w, matrix_mem_size, hipMemcpyDeviceToHost));

    hipFree(d_w);
    hipFree(d_u);
}

#undef M
#undef N