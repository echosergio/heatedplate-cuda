
#include <hip/hip_runtime.h>
#define M 50
#define N 50

#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

static void HandleError(hipError_t err, const char *file, int line)
{
    if (err != hipSuccess)
    {
        fprintf(stderr, "%s in %s at line %d\n", hipGetErrorString(err), file, line);
        exit(EXIT_FAILURE);
    }
}

#define CHECK_CUDA_ERROR(msg) (checkCUDAError(msg, __FILE__, __LINE__))

static void checkCUDAError(const char *msg, const char *file, int line)
{
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Cuda error: %s: %s. In %s at line %d\n", msg, hipGetErrorString(err), file, line);
        exit(EXIT_FAILURE);
    }
}

__global__ void copy_grid(double *d_w, double *d_u)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;

    if (x > 0 && y > 0 && x < M - 1 && y < N - 1)
    {
        int index = x + y * N;
        d_u[index] = d_w[index];
        __syncthreads();
    }

    return;
}

// CUDA kernel to perform the reduction in parallel on the GPU
//! @param g_idata  input data in global memory
//                  result is expected in index 0 of g_idata
//! @param n        input number of elements to scan from input data
__global__ void calculate_solution(double *d_w, double *d_u, double epsilon, double diff)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;

    if (x > 0 && y > 0 && x < M - 1 && y < N - 1)
    {
        int index = x + y * N;
        int left = (x - 1) + y * N;
        int right = (x + 1) + y * N;
        int top = x + (y -1) * N;
        int bottom = x + (y + 1) * N;
        
        d_w[index] = (d_u[left] + d_u[right] + d_u[top] + d_u[bottom]) / 4.0;
        __syncthreads();
    }

    return;
}

void calculate_solution_kernel(double w[M][N], double epsilon, double diff)
{
    const unsigned int matrix_mem_size = sizeof(double) * M * N;

    double *d_w = (double *)malloc(matrix_mem_size);
    double *d_u = (double *)malloc(matrix_mem_size);

    // Memory allocation on device side
    HANDLE_ERROR(hipMalloc((void **)&d_w, matrix_mem_size));
    HANDLE_ERROR(hipMalloc((void **)&d_u, matrix_mem_size));

    // Copy from host memory to device memory
    HANDLE_ERROR(hipMemcpy(d_w, w, matrix_mem_size, hipMemcpyHostToDevice));

    // Dimensions for a 2D matrix with max size 512
    dim3 dimGrid(16, 16); // 256 blocks 
    dim3 dimBlock(32, 32); // 1024 threads

    copy_grid<<<dimGrid,dimBlock>>>(d_w, d_u);

    // Invoke the kernel
    calculate_solution<<<dimGrid,dimBlock>>>(d_w, d_u, epsilon, diff);

    hipDeviceSynchronize();
    CHECK_CUDA_ERROR("kernel invocation");

    // Copy from device memory back to host memory
    HANDLE_ERROR(hipMemcpy(w, d_w, matrix_mem_size, hipMemcpyDeviceToHost));

    hipFree(d_w);
    hipFree(d_u);
}

#undef M
#undef N